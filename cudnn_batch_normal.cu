#include "cudnn_func.hpp"

template <typename Dtype>
syshen_batchnorm<Dtype>::syshen_batchnorm(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&scale_bias_desc));
	if (handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}
}

template <typename Dtype>
syshen_batchnorm<Dtype>::~syshen_batchnorm() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(scale_bias_desc));
	if (set_cudnn_handle) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(handle_t));
	}
}

template <typename Dtype>
void syshen_batchnorm<Dtype>::SetUp() {
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, 
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, 
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnDeriveBNTensorDescriptor(scale_bias_desc, input_desc, mode_));
	CHECK_CUDNN_ERROR(hipdnnDeriveBNTensorDescriptor(scale_bias_desc, output_desc, mode_));
	mode_ = HIPDNN_BATCHNORM_SPATIAL;
}

template <typename Dtype>
void syshen_batchnorm<Dtype>::Forward(Dtype *x, Dtype *y, Dtype *global_mean,
	Dtype *global_var, Dtype *bnScale, Dtype *bnBias) {
	Dtype alpha = Dtype(1.0);
	Dtype beta = Dype(0.0);
	CHECK_CUDNN_ERROR(hipdnnBatchNormalizationForwardInference(handle_t,
		mode_,
		&alpha, 
		&beta, 
		input_desc,
		x, 
		output_desc,
		y,
		scale_bias_desc,
		bnScale,
		bnBias,
		global_mean,
		global_var,
		HIPDNN_BN_MIN_EPSILON));
}