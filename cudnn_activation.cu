#include "cudnn_func.hpp"

template <typename Dtype>
syshen_activation<Dtype>::syshen_activation(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateActivationDescriptor(&act_desc));
	if (!handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}
}

template <typename Dtype>
syshen_activation<Dtype>::~syshen_activation() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyActivationDescriptor(&act_desc));
	if (set_cudnn_handle_t) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(&handle_t));
	}
}

template <typename Dtype>
void syshen_activation<Dtype>::SetUp() {
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetActivationDescriptor(act_desc, mode_, HIPDNN_NOT_PROPAGATE_NAN, Dtype(0)));
}

template <typename Dtype>
void syshen_activation<Dtype>::Forward(Dtype *x, Dtype *y) {
	Dtype alpha = 1.0f;
	Dtype beta = 0;
	CHECK_CUDNN_ERROR(hipdnnActivationForward(handle_t, act_desc,
		&alpha, input_desc, x, &beta, output_desc, y));
}