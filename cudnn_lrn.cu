#include "cudnn_func.hpp"

template <typename Dtype>
syshen_lrn<Dtype>::syshen_lrn(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateLRNDescriptor(&lrn_desc));
	if (handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}
}

template <typename Dtype>
syshen_lrn<Dtype>::~syshen_lrn() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyLRNDescriptor(lrn_desc));
	if (set_cudnn_handle) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(handle_t));
	}
}

template <typename Dtype>
void syshen_lrn<Dtype>::SetUp() {
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetLRNDescriptor(lrn_desc, lrnN, lrnAlpha, lrnBeta, lrnK));
}

template <typename Dtype>
void syshen_lrn<Dtype>::Forward(Dtype *x, Dtype *y) {
	Dtype alpha = Dtype(1.0);
	Dtype beta = Dtype(0.0);
	hipdnnLRNCrossChannelForward(handle_t, lrn_desc, 
		HIPDNN_LRN_CROSS_CHANNEL, &alpha, input_desc, x, &beta, output_desc, y);
}