#include "hip/hip_runtime.h"

#include "tools.h"
__global__ void checkIndex() {
	printf("threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\r\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

void test_check() {
	int nElem = 6;
	dim3 block(3);
	dim3 grid((nElem + block.x - 1) / block.x);

	printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
	printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

	checkIndex << <grid, block >> > ();

	hipDeviceReset();

}

void test_change_block() {
	int nElem = 1024;
	dim3 block(1024);
	dim3 grid((nElem + block.x - 1) / block.x);
	printf("grid.x %d block.x %d\n", grid.x, block.x);


	block.x = 512;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x %d block.x %d\n", grid.x, block.x);

	block.x = 256;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x %d block.x %d\n", grid.x, block.x);

	block.x = 128;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x %d block.x %d\n", grid.x, block.x);
}


const char* cublasGetErrorString(hipblasStatus_t error) {
	switch (error) {
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
	}
	return "Unknown cublas status";
}


