#include "cudnn_func.hpp"

template <typename Dtype>
syshen_pooling<Dtype>::syshen_pooling(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreatePoolingDescriptor(&poolingDesc));
	if (!handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}
}

template <typename Dtype>
syshen_pooling<Dtype>::~syshen_pooling() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyPoolingDescriptor(poolingDesc));
	if (set_cudnn_handle) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(handle_t));
	}
}

template <typename Dtype>
void syshen_pooling<Dtype>::SetUp(hipdnnPoolingMode_t mode) {
	mode_ = mode;
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channels, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channels, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetPooling2dDescriptor(poolingDesc, mode_,
		HIPDNN_NOT_PROPAGATE_NAN, kernel_h, kernel_w, 
		pad_h, pad_w, stride_h, stride_w));
}

template <typename Dtype>
void syshen_pooling<Dtype>::Forward(Dtype *x, Dtype *y) {
	Dtype alpha = 1.0f;
	Dtype beta = 0;
	CHECK_CUDNN_ERROR(hipdnnPoolingForward(handle_t, poolingDesc,
		&alpha, input_desc, x, &beta, output_desc, y));
}