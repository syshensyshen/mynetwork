#include "cudnn_func.hpp"

template <typename Dtype>
syshen_deconvolution<Dtype>::syshen_deconvolution(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateFilterDescriptor(&filter_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateConvolutionDescriptor(&conv_desc));
	if (has_bias) {
		CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&bias));
	}
	if (!handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}

	if (use_stream) {
		CHECK_CUDA_ERROR(hipStreamCreate(&stream));
		CHECK_CUDA_ERROR(hipEventCreate(&start));
	}
	batch = 1;
	in_channels = 1;
	stride_h = 1;
	stride_w = 1;
	pad_h = 1;
	pad_w = 1;
	dilation_h = 1;
	dilation_w = 1;
	kernel_h = 1;
	kernel_w = 1;
}

template <typename Dtype>
syshen_deconvolution<Dtype>::~syshen_deconvolution() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyFilterDescriptor(filter_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyConvolutionDescriptor(conv_desc));
	if (has_bias) {
		CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(bias));
	}
	if (set_cudnn_handle) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(handle_t));
	}

	if (use_stream) {
		CHECK_CUDA_ERROR(hipStreamDestroy(stream));
		//CHECK_CUDA_ERROR(hipEventDestroy(strat));
	}
}

template<typename Dtype>
void syshen_deconvolution<Dtype>::SetUp() {
	int nStride = in_channels * in_h * in_w;
	int cStride = in_h * in_w;

	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptorEx(
		input_desc,
		hipdnnDataType_t::HIPDNN_DATA_FLOAT,
		batch,
		in_channels,
		in_h, in_w, nStride, cStride, in_w, 1));

	CHECK_CUDNN_ERROR(hipdnnSetFilter4dDescriptor(
		filter_desc,
		hipdnnDataType_t::HIPDNN_DATA_FLOAT,
		HIPDNN_TENSOR_NCHW,
		out_channels, in_channels, kernel_h, kernel_w));

	CHECK_CUDNN_ERROR(hipdnnSetConvolution2dDescriptor(
		conv_desc, pad_h, pad_w, stride_h,
		stride_w, dilation_h, dilation_w,
		HIPDNN_CROSS_CORRELATION, hipdnnDataType_t::HIPDNN_DATA_FLOAT));

	/*CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
	conv_desc, input_desc, filter_dsec,
	&out_batch, &out_channels, &out_h, &out_w));*/

	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(
		output_desc, HIPDNN_TENSOR_NCHW,
		hipdnnDataType_t::HIPDNN_DATA_FLOAT,
		out_batch, out_channels, out_h, out_w));

	/*CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardFilterAlgorithm(
		handle_t, input_desc, filter_desc,
		conv_desc, output_desc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
		0, &algo));

	CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
		handle_t, input_desc, output_desc,
		conv_desc, filter_desc, algo, &workSpaceSize));*/
	CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardDataAlgorithm(
		handle_t, filter_desc, input_desc, conv_desc,
		output_desc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algo));
	CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardDataWorkspaceSize(
		handle_t, filter_desc, input_desc, conv_desc, 
		output_desc, algo, &workSpaceSize));
	if (0 != workSpaceSize)
		CHECK_CUDA_ERROR(hipMalloc((void**)&workSpace, workSpaceSize));

	if (has_bias) {
		hipdnnSetTensor4dDescriptor(bias, HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, out_batch, out_channels, 1, 1);
	}
}

template<typename Dtype>
void syshen_deconvolution<Dtype>::Forward(Dtype *input, Dtype *output, Dtype *weights, Dtype *bias_weights) {
	Dtype conv_alpha = 1.0f;
	Dtype conv_beta = 0;
	CHECK_CUDNN_ERROR(hipdnnConvolutionBackwardData(
		handle_t, &conv_alpha, filter_desc, weights, 
		input_desc, input, conv_desc, algo, workSpace,
		workSpaceSize, &conv_beta, output_desc, output));
	if (has_bias) {
		hipdnnAddTensor(handle_t, &conv_alpha, bias, bias_weights, &conv_alpha, output_desc, output);
	}
}