#include "cudnn_func.hpp"

template <typename Dtype>
syshen_softmax<Dtype>::syshen_softmax(hipdnnHandle_t handle_) {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	if (!handle_) {
		CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
		set_cudnn_handle = true;
	}
	else {
		handle_t = handle_;
		set_cudnn_handle = false;
	}
}

template <typename Dtype>
syshen_softmax<Dtype>::~syshen_softmax() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(&output_desc));

	if (set_cudnn_handle_t) {
		CHECK_CUDNN_ERROR(hipdnnDestroy(&handle_t));
	}
}

template <typename Dtype>
void syshen_softmax<Dtype>::SetUp() {
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT, batch, channles, height, width));
}

template <typename Dtype>
void syshen_softmax<Dtype>::Forward(Dtype *x, Dtype *y) {
	Dtype alpha = Dtype(1.0);
	Dtype beta = Dtype(0.0);
	mode_ = HIPDNN_SOFTMAX_MODE_CHANNEL;
	CHECK_CUDNN_ERROR(hipdnnSoftmaxForward(handle_t, HIPDNN_SOFTMAX_ACCURATE, mode_,
		&alpha, input_desc, x, &beta, output_desc, y));
}